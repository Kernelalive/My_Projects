/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005 

 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;		//edw einai to pou tha paei to filtro gi auto elegxei apo katw kai an to d einia ektos oriwn eikonas

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}

__global__ void convolutionRowGPU(float *h_Dst, float *h_Src, float *h_Filter, int imageW, int imageH, int filterR){

  int k,i,j;
  float sum = 0;
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;

   for (i = 0; i < imageW ; i++){
    for(j = 0; j < imageW; j++){
      printf("%f ", h_Src[i * imageW + j]); //dn kserw an auto to printf eprepe na mou typwnei swsta apotelesmata episis ta stelnei sth Cpu kai meta sto strout h apo thn Gpu?? 
    }
    printf("\n"); 
  }
  for (k = -filterR; k <= filterR; k++) {
    int d = ix + k;    //edw einai to pou tha paei to filtro gi auto elegxei apo katw kai an to d einia ektos oriwn eikonas
    sum += h_Src[iy * imageH + d] * h_Filter[filterR - k];
    h_Dst[iy * imageW + ix] = sum;
    printf("%f", h_Dst[iy * imageW + ix]);
  }
}


__global__ void convolutionColumnGPU(float *h_Dst, float *h_Src, float *h_Filter, int imageW, int imageH, int filterR){
	int k;
	float sum = 0;
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
  for (k = -filterR; k <= filterR; k++) {
    int d = iy + k;
    sum += h_Src[d * imageW + ix] * h_Filter[filterR - k];
    h_Dst[iy * imageW + ix] = sum;
  }
}



////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *dev_Filter,
    *dev_Input,
    *dev_Buffer,
    *dev_OutputGPU,
    *dev_Padding,
    *h_apotelesmata,
    *h_Padding;


    int imageW;
    int imageH;
    int new_image_sizeH, new_image_sizeW;
    int block_Dim, grid_Dim;
    unsigned int i;
    unsigned int j;
    //FILE * fp;
    //fp = fopen("Max_accuracy.txt", "a");
    

    printf("Enter filter radius : ");
    scanf("%d", &filter_radius);
    //fprintf(fp,"for filter radius %d  ", filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;


    new_image_sizeW = imageW + 2 * filter_radius;   //to megethos tou padded  pinaka (h_Padding)
    new_image_sizeH = imageH + 2 * filter_radius;


    if (new_image_sizeW > 32){
      block_Dim = 32;
      grid_Dim = imageW / 32;
    }
    else{
      block_Dim = new_image_sizeW;
      grid_Dim = 1;
    }



    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Padding = (float *)malloc(new_image_sizeW * new_image_sizeH * sizeof(float));
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_apotelesmata = (float *)malloc(imageW * imageH * sizeof(float)); //gia na epistrepsw to apotelesma apo thn GPU

    if ((h_Filter == 0 )||(h_Input == 0) || (h_Buffer == 0) || (h_OutputCPU == 0) || (h_Padding == 0)){
        printf("Failure in memory allocation\n");
        exit (0);
    }
    //Device allocation
    int size = imageW * imageH * sizeof(float);
    hipMalloc((void**)&dev_Padding, new_image_sizeW * new_image_sizeH * sizeof(float));
    hipMalloc((void**)&dev_Filter, FILTER_LENGTH * sizeof(float));
    hipMalloc((void**)&dev_Input, size);
    hipMalloc((void**)&dev_Buffer, new_image_sizeW * new_image_sizeH * sizeof(float));//allazei kai to size tou buffer afou to prwto convolution apothikeuei to apotelesma edw 
    hipMalloc((void**)&dev_OutputGPU, size);

    if ((dev_Filter == 0 )||(dev_Input == 0) || (dev_Buffer == 0) || (dev_OutputGPU == 0) || (dev_Padding == 0)){
        printf("Failure in Gpu memory allocation\n");
        exit (0);
    }

    //arxikopoihsh gia kathe endexomeno
    hipMemset(dev_Padding, 0 , new_image_sizeW * new_image_sizeH * sizeof(float));
    hipMemset(dev_Filter, 0 , FILTER_LENGTH * sizeof(float));
    hipMemset(dev_Input, 0, size);
    hipMemset(dev_Buffer, 0, new_image_sizeW * new_image_sizeH * sizeof(float));
    hipMemset(dev_OutputGPU, 0, size);

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }
    //for (i = 0; i < imageW ; i++){
      //for(j = 0; j < imageH; j++){
       // printf("%f ",h_Input[i * imageH + j]); 
      //}
     // printf("\n"); 
    //}
    
    //Mhdenizw ton padded pinaka  
    for (i = 0; i < new_image_sizeW; i++){
      for (j = 0; j< new_image_sizeH; j++){
        h_Padding[i * new_image_sizeW + j] = 0;
      }
    }

    //Enopoiw tous 2 pinakes wste na ftiaxtei o telikos padded pinakas
    for (i = 0; i < imageW; i++){
      for (j = 0; j < imageH; j++){
        h_Padding[(i + filter_radius) * new_image_sizeW + (j+filter_radius)] = h_Input[i * imageH + j];

      }
    }


    for (i = 0; i < new_image_sizeW ; i++){
      for(j = 0; j < new_image_sizeH; j++){
        printf("%f ",h_Padding[i * new_image_sizeH + j]); 
      }
      printf("\n"); 
    }

    //memcpy afou gemisoun oi pinakes(eikones)
    hipMemcpy(dev_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_Padding, h_Padding, new_image_sizeW * new_image_sizeH * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimGrid(grid_Dim, grid_Dim);
    dim3 dimBlock(block_Dim, block_Dim);
    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

    for (i = 0; i < imageW * imageH; i++) {    // stamataw tis ektypwseis gt dn pairnei apeiro xrono mexri na tis deiksei oles
      printf("%f\n",h_OutputCPU[i]);
    }

    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas 

    printf("GPU computation...\n");
    convolutionRowGPU<<<dimGrid, dimBlock>>>(dev_Buffer, dev_Padding, dev_Filter, new_image_sizeW, imageW, filter_radius);

    // check for error dn vgainei gia na dw gt xtypaei to 2o error 
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
    }
    
    convolutionColumnGPU<<<dimGrid, dimBlock>>>(dev_OutputGPU, dev_Buffer, dev_Filter, imageW, imageH, filter_radius);

    // check for error
    hipError_t error1 = hipGetLastError();
    if(error1 != hipSuccess){
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error1));
    }
    
    hipMemcpy(h_apotelesmata, dev_OutputGPU , size, hipMemcpyDeviceToHost);

    for (i = 0; i < imageW * imageH; i++) {
      printf("%f\n",h_apotelesmata[i]);
    }
    

    float diafora = 0;
    float Max_acc = -1;
    for (i = 0; i < imageW * imageH; i++) {
      diafora = h_OutputCPU[i] - h_apotelesmata[i];
      diafora = ABS(diafora);
      if (diafora > Max_acc){
      	Max_acc = diafora;
      }
    }

    printf ("to grid einia %d kai to block einia %d\n", grid_Dim, block_Dim);
    printf ("The Max_accuracy is %f\n", Max_acc);
    //fprintf(fp,"the Max accuracy is    %32f\n",Max_acc);
           
    if(Max_acc > accuracy){
    	printf("CPU and GPU results don't match and program is gonna terminate\n");
      	exit(0);
    }
    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    free(h_Padding);
    free(h_apotelesmata);
    hipFree(dev_Padding);
    hipFree(dev_Filter);
    hipFree(dev_Input);
    hipFree(dev_Buffer);
    hipFree(dev_OutputGPU);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}

