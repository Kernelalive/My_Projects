#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

PGM_IMG contrast_enhancement_g(PGM_IMG img_in)
{
    PGM_IMG result, img_in_Gpu, result_in_Gpu;
    int * hist_in_Gpu, * dev_lut;
    int block_Dim, grid_Dimx, grid_Dimy;


    result.w = img_in.w;
    result.h = img_in.h;
	

    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    hipMalloc((void **)&result_in_Gpu.img, result.w * result.h * sizeof(unsigned char));
    hipMalloc((void **)&img_in_Gpu.img, result.w * result.h * sizeof(unsigned char));
    hipMalloc((void **)&hist_in_Gpu, 256 * sizeof(int));
    hipMalloc((void**)&dev_lut, 256 * sizeof(int));
    if ((result_in_Gpu.img == 0 )||(hist_in_Gpu == 0) || (img_in_Gpu.img == 0) || (dev_lut == 0)){
        printf("Failure in Gpu memory allocation\n");
        exit (0);
    }



    hipMemcpy(result_in_Gpu.img, result.img,  result.w * result.h * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(img_in_Gpu.img, img_in.img,  result.w * result.h * sizeof(unsigned char), hipMemcpyHostToDevice);



    if(result.w > 32){     
        block_Dim = 32;
         grid_Dimx = (result.w / 32) + 1;
         grid_Dimy = (result.w / 32) + 1;
    }
    else{
        block_Dim = result.w;
        grid_Dimx = 1;
        grid_Dimy = 1;
    }
    
    dim3 dimBlock(block_Dim,block_Dim);
    dim3 dimGrid(grid_Dimx,grid_Dimy);

    calc_histogram<<<dimGrid,dimBlock>>>(hist_in_Gpu, img_in_Gpu.img,  result.w*result.h, 256);

    hipDeviceSynchronize();
    hipError_t error=hipGetLastError();
    if(error!=hipSuccess){
        printf("Cuda Error in calc_histogram:%s\n",hipGetErrorString(error));
    }

    calc_lut<<<1,256>>>(dev_lut, hist_in_Gpu, result.w*result.h, 256);

    hipDeviceSynchronize();
    hipError_t error1=hipGetLastError();
    if(error1!=hipSuccess){
    printf("Cuda Error in calc_lut:%s\n",hipGetErrorString(error1));
    }

    calc_output<<<dimGrid,dimBlock>>>(result_in_Gpu.img, img_in_Gpu.img, dev_lut, result.w*result.h);

    hipDeviceSynchronize();
    hipError_t error2=hipGetLastError();
    if(error2!=hipSuccess){
        printf("Cuda Error in calc_output:%s\n",hipGetErrorString(error2));
    }

    hipMemcpy(result.img, result_in_Gpu.img,  result.w * result.h * sizeof(unsigned char), hipMemcpyDeviceToHost);
    return result;
}



















