#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"
#define STREAM_NUM 2

void run_cpu_gray_test(PGM_IMG img_in, char *out_filename);

int main(int argc, char *argv[]){
    PGM_IMG img_ibuf_g;

	if (argc != 3) {
		printf("Run with input file name and output file name as arguments\n");
		exit(1);
	}
	
    printf("Running contrast enhancement for gray-scale images.\n");
    img_ibuf_g = read_pgm(argv[1]);
    run_cpu_gray_test(img_ibuf_g, argv[2]);
    free_pgm(img_ibuf_g);

	return 0;
}



void run_cpu_gray_test(PGM_IMG img_in, char *out_filename)
{
    unsigned int timer = 0;
    PGM_IMG img_obuf;
    float gpu_time;
    hipEvent_t start;
    hipEvent_t stop;



    const int nstreams = STREAM_NUM;
    int img_Size = img_in.w * img_in.h;
    int streamSize = img_Size / nstreams;
    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for(int i=0; i<nstreams; i++) {
        hipStreamCreate(&(streams[i]));
    }

    printf("Starting GPU processing...\n");
    hipEventRecord(start,0);

    //img_obuf = contrast_enhancement_g(img_in);
      PGM_IMG result, img_in_Gpu, result_in_Gpu;
    int * hist_in_Gpu, * dev_lut;
    int block_Dim, grid_Dimx, grid_Dimy;
 


    result.w = img_in.w;
    result.h = img_in.h;
    if(result.w > 32){     
        block_Dim = 32;
         grid_Dimx = (result.w / 32) + 1;
         grid_Dimy = (result.w / 32) + 1;
    }
    else{
        block_Dim = result.w;
        grid_Dimx = 1;
        grid_Dimy = 1;
    }

    dim3 dimBlock(block_Dim,block_Dim);
    dim3 dimGrid(grid_Dimx,grid_Dimy);

    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    hipMalloc((void **)&result_in_Gpu.img, result.w * result.h * sizeof(unsigned char));
    hipMalloc((void **)&img_in_Gpu.img, result.w * result.h * sizeof(unsigned char));
    hipMalloc((void **)&hist_in_Gpu, 256 * sizeof(int));
    hipMalloc((void**)&dev_lut, 256 * sizeof(int));
    if ((result_in_Gpu.img == 0 )||(hist_in_Gpu == 0) || (img_in_Gpu.img == 0) || (dev_lut == 0)){
        printf("Failure in Gpu memory allocation\n");
        exit (0);
    }



    //hipMemcpy(result_in_Gpu.img, result.img,  result.w * result.h * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(img_in_Gpu.img, img_in.img,  result.w * result.h * sizeof(unsigned char), hipMemcpyHostToDevice);
    for (int i=0; i<nstreams; i++) {
        int offset = i * streamSize;
        int offset1 = (i + 1) * streamSize;
        hipMemcpyAsync(&result_in_Gpu.img[offset], &result.img[offset], streamSize * sizeof(unsigned char),
                                      hipMemcpyHostToDevice, streams[i]);
        calc_histogram<<<256,256,0,streams[i]>>>(img_in_Gpu.img, offset, offset1);
    }


    hipDeviceSynchronize();
    hipError_t error=hipGetLastError();
    if(error!=hipSuccess){
        printf("Cuda Error in calc_histogram:%s\n",hipGetErrorString(error));
    }

    calc_lut<<<1,256>>>(dev_lut, result.w*result.h);

    hipDeviceSynchronize();
    hipError_t error1=hipGetLastError();
    if(error1!=hipSuccess){
    printf("Cuda Error in calc_lut:%s\n",hipGetErrorString(error1));
    }
    //for (int i=0; i<nstreams; i++) {
     //  int offset = i * streamSize;
     //   int offset1 = (i + 1) * streamSize;
      //  calc_output<<<256,256,0,streams[i]>>>(img_in_Gpu.img, offset, offset1);
    //}


    calc_output<<<dimGrid,dimBlock>>>(result_in_Gpu.img, img_in_Gpu.img, dev_lut, result.w*result.h);

    hipDeviceSynchronize();
    hipError_t error2=hipGetLastError();
    if(error2!=hipSuccess){
        printf("Cuda Error in calc_output:%s\n",hipGetErrorString(error2));
    }

    //for (int i=0; i<nstreams; i++) {
      //  int offset = i * streamSize;
      //  int offset1 = (i + 1) * streamSize;
      //  hipMemcpyAsync(&result.img[offset], &result_in_Gpu.img[offset],streamSize * sizeof(unsigned char),hipMemcpyDeviceToHost, streams[i]);
    //}





    hipMemcpy(result.img, result_in_Gpu.img,  result.w * result.h * sizeof(unsigned char), hipMemcpyDeviceToHost);
    img_obuf = result;




    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time,start,stop);

    printf("GPU time used in seconds is %f\n", gpu_time );

    write_pgm(img_obuf, out_filename);
    free_pgm(img_obuf);
}


PGM_IMG read_pgm(const char * path){
    FILE * in_file;
    char sbuf[256];
    
    
    PGM_IMG result;
    int v_max;//, i;
    in_file = fopen(path, "r");
    if (in_file == NULL){
        printf("Input file not found!\n");
        exit(1);
    }
    
    fscanf(in_file, "%s", sbuf); /*Skip the magic number*/
    fscanf(in_file, "%d",&result.w);
    fscanf(in_file, "%d",&result.h);
    fscanf(in_file, "%d\n",&v_max);
    printf("Image size: %d x %d\n", result.w, result.h);
    

    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));

        
    fread(result.img,sizeof(unsigned char), result.w*result.h, in_file);    
    fclose(in_file);
    
    return result;
}

void write_pgm(PGM_IMG img, const char * path){
    FILE * out_file;
    out_file = fopen(path, "wb");
    fprintf(out_file, "P5\n");
    fprintf(out_file, "%d %d\n255\n",img.w, img.h);
    fwrite(img.img,sizeof(unsigned char), img.w*img.h, out_file);
    fclose(out_file);
}

void free_pgm(PGM_IMG img)
{
    free(img.img);
}

