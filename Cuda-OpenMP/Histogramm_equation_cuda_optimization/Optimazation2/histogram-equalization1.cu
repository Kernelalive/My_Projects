#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

#define COLORS 256

__device__ int glb_hist[COLORS];


__global__ void calc_histogram(unsigned char * img_in, int offset_start, int offset_end){
     int ix = blockIdx.x * blockDim.x + threadIdx.x;;
    const int gridW = gridDim.x * blockDim.x;
    int Row, pos;

    __shared__ int hist[COLORS];

    if (threadIdx.x < COLORS) {
        hist[threadIdx.x] = 0;
    }
    __syncthreads();

    int RowNum = (offset_end - offset_start) / gridW;
    int extras = (offset_end - offset_start) % gridW;
    


    for (Row=0; Row<RowNum; Row++) {
        pos = Row * gridW + ix;
        atomicAdd(&hist[img_in[pos + offset_start]],1);
    }
    if (extras && ix < extras) {
        pos = Row * gridW + ix;
        atomicAdd(&hist[img_in[pos + offset_start]],1);
    }
    __syncthreads();
    if (threadIdx.x < COLORS) {
        atomicAdd(&glb_hist[threadIdx.x],hist[threadIdx.x]);
    }
}
 //int *lut = (int *)malloc(sizeof(int)*nbr_bin);
__global__ void calc_lut(int *lut, int img_size){ 

    
    __shared__ int shared_hist[COLORS];
    shared_hist[threadIdx.x] = glb_hist[threadIdx.x];
    __syncthreads();
    __shared__  int cdf[COLORS];
    __syncthreads();

    int i, min, d; 
    //int cdf = 0;
    min = 0;
    i = 0;

    while(min == 0){
        min = shared_hist[i++];
    }
    d = img_size - min;
    for (unsigned int stride = 1; stride <= threadIdx.x; stride *= 2) {
        __syncthreads();
        shared_hist[threadIdx.x] += shared_hist[threadIdx.x-stride];
    }
    cdf[threadIdx.x] = shared_hist[threadIdx.x];
    //printf("cdf = %d\n",cdf);
    __syncthreads();
    
	

    //for(i = 0; i <= threadIdx.x; i ++){	//tha mporouse na ginei me prefix sum san veltistoipohsh FIXME
      //  cdf += shared_hist[i];
      //  lut[i] = (cdf - min)*(nbr_bin - 1)/d;        
    //}
    //printf("cdf = %d\n",cdf);
    

    lut[threadIdx.x] = (int)(((float)cdf[threadIdx.x] - min)*255/d + 0.5);
    if(lut[threadIdx.x] < 0){
        lut[threadIdx.x] = 0;
    }
}

__global__ void calc_output(unsigned char * img_out, unsigned char * img_in, int * lut, int img_size){
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    const int gridW = gridDim.x * blockDim.x;
    int img_position1 = iy * gridW + ix; //thesh mesa sthn eikona synarthsh tou gridW 
    __syncthreads();
    if(img_position1 < img_size){
        if(lut[img_in[img_position1]] > 255){
            img_out[img_position1] = 255;
        }
        else{
            img_out[img_position1] = (unsigned char)lut[img_in[img_position1]];
        }
    }
        
    
}
