/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005 

 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;		//edw einai to pou tha paei to filtro gi auto elegxei apo katw kai an to d einia ektos oriwn eikonas

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}

__global__ void convolutionRowGPU(float *h_Dst, float *h_Src, float *h_Filter, int imageW, int imageH, int filterR){

  int k;
  float sum = 0;
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  for (k = -filterR; k <= filterR; k++) {
    int d = ix + k;    //edw einai to pou tha paei to filtro gi auto elegxei apo katw kai an to d einia ektos oriwn eikonas
    if (d >= 0 && d < imageW) {
      sum += h_Src[iy * imageW + d] * h_Filter[filterR - k];
    }
    h_Dst[iy * imageW + ix] = sum;
  }
}


__global__ void convolutionColumnGPU(float *h_Dst, float *h_Src, float *h_Filter, int imageW, int imageH, int filterR){
	int k;
	float sum = 0;
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
  for (k = -filterR; k <= filterR; k++) {
    int d = iy + k;
    if (d >= 0 && d < imageH) {
      sum += h_Src[d * imageW + ix] * h_Filter[filterR - k];
    }
    h_Dst[iy * imageW + ix] = sum;
  }
}



////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *dev_Filter,
    *dev_Input,
    *dev_Buffer,
    *dev_OutputGPU,
    *h_apotelesmata;


    int imageW;
    int imageH;
    int block_Dim, grid_Dim;
    unsigned int i;
    //FILE * fp;
    //fp = fopen("Max_accuracy.txt", "a");
    

    printf("Enter filter radius : ");
    scanf("%d", &filter_radius);
    //fprintf(fp,"for filter radius %d  ", filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    if (imageW > 32){
      block_Dim = 32;
      grid_Dim = imageW / 32;
    }
    else{
      block_Dim = imageW;
      grid_Dim = 1;
    }

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_apotelesmata = (float *)malloc(imageW * imageH * sizeof(float)); //gia na epistrepsw to apotelesma apo thn GPU

    if ((h_Filter == 0 )||(h_Input == 0) || (h_Buffer == 0) || (h_OutputCPU == 0)){
        printf("Failure in memory allocation\n");
        exit (0);
    }
    //Device allocation
    int size = imageW * imageH * sizeof(float);
    hipMalloc((void**)&dev_Filter, FILTER_LENGTH * sizeof(float));
    hipMalloc((void**)&dev_Input, size);
    hipMalloc((void**)&dev_Buffer, size);
    hipMalloc((void**)&dev_OutputGPU, size);

    if ((dev_Filter == 0 )||(dev_Input == 0) || (dev_Buffer == 0) || (dev_OutputGPU == 0)){
        printf("Failure in Gpu memory allocation\n");
        exit (0);
    }

    //arxikopoihsh gia kathe endexomeno
    hipMemset(dev_Filter, 0 , FILTER_LENGTH * sizeof(float));
    hipMemset(dev_Input, 0, size);
    hipMemset(dev_Buffer, 0, size);
    hipMemset(dev_OutputGPU, 0, size);

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }
    
    //memcpy afou gemisoun oi pinakes(eikones)
    hipMemcpy(dev_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_Input, h_Input, size, hipMemcpyHostToDevice);

    dim3 dimGrid(grid_Dim, grid_Dim);
    dim3 dimBlock(block_Dim, block_Dim);
    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

    //for (i = 0; i < imageW * imageH; i++) {    // stamataw tis ektypwseis gt dn pairnei apeiro xrono mexri na tis deiksei oles
    //  printf("%f\n",h_OutputCPU[i]);
   // }

    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas 

    printf("GPU computation...\n");
    convolutionRowGPU<<<dimGrid, dimBlock>>>(dev_Buffer, dev_Input, dev_Filter, imageW, imageH, filter_radius);

    // check for error dn vgainei gia na dw gt xtypaei to 2o error 
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
    }
    
    convolutionColumnGPU<<<dimGrid, dimBlock>>>(dev_OutputGPU, dev_Buffer, dev_Filter, imageW, imageH, filter_radius);

    // check for error
    hipError_t error1 = hipGetLastError();
    if(error1 != hipSuccess){
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error1));
    }
    
    hipMemcpy(h_apotelesmata, dev_OutputGPU , size, hipMemcpyDeviceToHost);

    //for (i = 0; i < imageW * imageH; i++) {
     // printf("%f\n",h_apotelesmata[i]);
    //}
    

    float diafora = 0;
    float Max_acc = -1;
    for (i = 0; i < imageW * imageH; i++) {
      diafora = h_OutputCPU[i] - h_apotelesmata[i];
      diafora = ABS(diafora);
      if (diafora > Max_acc){
      	Max_acc = diafora;
      }
    }

    printf ("to grid einia %d kai to block einia %d\n", grid_Dim, block_Dim);
    printf ("The Max_accuracy is %f\n", Max_acc);
    //fprintf(fp,"the Max accuracy is    %32f\n",Max_acc);
           
    if(Max_acc > accuracy){
    	printf("CPU and GPU results don't match and program is gonna terminate\n");
      	exit(0);
    }
    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    free(h_apotelesmata);
    hipFree(dev_Filter);
    hipFree(dev_Input);
    hipFree(dev_Buffer);
    hipFree(dev_OutputGPU);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}

