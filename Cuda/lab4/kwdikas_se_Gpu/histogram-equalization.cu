#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"


__global__ void calc_histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin){
    
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    const int gridW = gridDim.x * blockDim.x;
    int img_position = iy * gridW + ix; //thesh mesa sthn eikona synarthsh tou gridW 

   // __shared__ hist_out[nbr_bin];

    if (img_position < nbr_bin) { //allagh tou img_position me threadIdx.x se auto to if anevazei poly to contrast
        hist_out[img_position] = 0;
    }

    __syncthreads();

    if(img_position < img_size){
        atomicAdd(&hist_out[img_in[img_position]],1);
    }
    __syncthreads();
}

  //int *lut = (int *)malloc(sizeof(int)*nbr_bin);
__global__ void calc_lut(int *lut, int * hist_in, int img_size, int nbr_bin){ 

    
    __shared__ int shared_hist[256];
    shared_hist[threadIdx.x] = hist_in[threadIdx.x];
    __syncthreads();

    int i, cdf, min, d; 
    cdf = 0;
    min = 0;
    i = 0;

    while(min == 0){
        min = shared_hist[i++];
    }
    d = img_size - min;
    for(i = 0; i <= threadIdx.x; i ++){	//tha mporouse na ginei me prefix sum san veltistoipohsh FIXME
        cdf += shared_hist[i];
        //lut[i] = (cdf - min)*(nbr_bin - 1)/d;        
    }

    lut[threadIdx.x] = (int)(((float)cdf - min)*255/d + 0.5);
    if(lut[threadIdx.x] < 0){
        lut[threadIdx.x] = 0;
    }
}
__global__ void calc_output(unsigned char * img_out, unsigned char * img_in, int * lut, int img_size){
    /* Get the result image */
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    const int gridW = gridDim.x * blockDim.x;
    int img_position1 = iy * gridW + ix; //thesh mesa sthn eikona synarthsh tou gridW 
    /* Construct the LUT by calculating the CDF */
    __syncthreads();
    if(img_position1 < img_size){
        if(lut[img_in[img_position1]] > 255){
            img_out[img_position1] = 255;
        }
        else{
            img_out[img_position1] = (unsigned char)lut[img_in[img_position1]];
        }
        
    }
}
