/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius ;

#define FILTER_LENGTH 	(2 * 16 + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005 
#define TILE_WIDTH  64
#define TILE_HEIGHT 64

 __constant__ float Filter_in_constant_mem[FILTER_LENGTH];  //pernaw to filtro sthn constant memory gt auto mono tha diavazetai

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;		//edw einai to pou tha paei to filtro gi auto elegxei apo katw kai an to d einia ektos oriwn eikonas

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}

__global__ void convolutionRowGPU(float *h_Dst, float *h_Src, float *h_Filter, int imageW, int imageH, int filterR){

  int k;
  float sum = 0;
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  for (k = -filterR; k <= filterR; k++) {
    int d = ix + k;    //edw einai to pou tha paei to filtro gi auto elegxei apo katw kai an to d einia ektos oriwn eikonas
    if (d >= 0 && d < imageW) {
      sum += h_Src[iy * imageW + d] * h_Filter[filterR - k];
    }
    h_Dst[iy * imageW + ix] = sum;
  }
}


__global__ void convolutionColumnGPU(float *h_Dst, float *h_Src, float *h_Filter, int imageW, int imageH, int filterR){
	int k;
	float sum = 0;
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
  for (k = -filterR; k <= filterR; k++) {
    int d = iy + k;
    if (d >= 0 && d < imageH) {
      sum += h_Src[d * imageW + ix] * h_Filter[filterR - k];
    }
    h_Dst[iy * imageW + ix] = sum;
  }
}



__global__ void convolutionTileRowGPU(float *h_Dst, float *h_Src, int imageW, int imageH, int filterR){


  __shared__ float tiled_Src[TILE_WIDTH][TILE_HEIGHT];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int k, i, l;

  int ix = bx * blockDim.x + tx;
  int iy = by * blockDim.y + ty;

  int Row = by * TILE_HEIGHT + ty;
  int Col = bx * TILE_WIDTH + tx;

  int p = TILE_WIDTH / blockDim.x;
  for (l = -(p / 2); l <= (p / 2); l++) { 
    int b = bx + l; 

    if (b >= 0 && b <= p ) {
      tiled_Src[ty][ix + l * blockDim.x] = h_Src[iy * imageW + ix + l * blockDim.x];
    }
  }
  __syncthreads(); //ola ta nhmata tou block exoun oloklhrwsei thn fortwsh ston tiled_Src


  //if(tiled_Src[ty][tx] != 0){
    //printf("tileSrC = %f\n",tiled_Src[ty][tx]);
  //}
  float sum = 0;

  for (k = -filterR; k <= filterR; k++) {
    int d = tx + k;  

    //if (d >= 0 && d < TILE_WIDTH ) {
      //sum += tiled_Src[ty][d] * Filter_in_constant_mem[filterR - k];
    //}
    if((ix + k > 0) && (ix + k < imageW)){//else if((ix + k > 0) && (ix + k < imageW)){
      sum += tiled_Src[ty] [(ix + k)] * Filter_in_constant_mem[filterR - k];//sum += tiled_Src[iy * imageW + (ix + k)] * Filter_in_constant_mem[filterR - k];
    }
    __syncthreads(); //gia na eksasfalisei oti ta nhmata tou block exoun teliwsei me ton tiled_Src

    h_Dst[iy * imageW + ix] = sum;
    __syncthreads();
  }
  __syncthreads();
}



__global__ void convolutionTileColumnGPU(float *h_Dst, float *h_Src, int imageW, int imageH, int filterR){


  __shared__ float tiled_Src[TILE_WIDTH][TILE_HEIGHT];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int k, l;

  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;

  int Row = by * TILE_HEIGHT + ty;
  int Col = bx * TILE_WIDTH + tx;


  int p = TILE_WIDTH / blockDim.y;
  for (l = -(p / 2); l <= (p / 2); l++) { 
    int b = bx + l; 

    if (b >= 0 && b <= p ) {
      tiled_Src[iy + l * blockDim.y][tx] = h_Src[(iy + l * blockDim.y) * imageW + ix];
    }
  }
  __syncthreads(); //ola ta nhmata tou block exoun oloklhrwsei thn fortwsh ston tiled_Src

  //if(tiled_Src[ty][tx] != 0){
    //printf("tileSrCol = %f\n",tiled_Src[ty][tx]);
  //}
  float sum = 0;

  for (k = -filterR; k <= filterR; k++) {
    int d = ty + k;  

    //if (d >= 0 && d < TILE_WIDTH ) {
    // sum += tiled_Src[d][tx] * Filter_in_constant_mem[filterR - k];
    //}
    if((iy + k > 0) && (iy + k < imageW)){//else if((iy + k > 0) && (iy + k < imageW)){
      sum += tiled_Src[(iy + k)][ix] * Filter_in_constant_mem[filterR - k];//sum += h_Src[(iy + k) * imageW + ix] * Filter_in_constant_mem[filterR - k];
    }
    __syncthreads(); //gia na eksasfalisei oti ta nhmata tou block exoun teliwsei me ton tiled_Src
    h_Dst[iy * imageW + ix] = sum;
    __syncthreads();
  }

__syncthreads();
}



////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *dev_Filter,
    *dev_Input,
    *dev_Buffer,
    *h_tiled,
    *dev_new_Buffer,
    *dev_OutputGPU,
    *dev_new_OutputGPU,
    *h_apotelesmata,
    *h_apotelesmata2;

    clock_t startCPU, startGPU, endGPU, endCPU;
    double cpu_time_used;

    float gpu_time_used_no_tile, gpu_time_used;

    hipEvent_t start, stop, start1, stop1;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&start1);
    hipEventCreate(&stop1);


    int imageW;
    int imageH;
    int block_Dim, grid_Dim;
    unsigned int i;
    FILE * fp;
    fp = fopen("Xronoi_Ekteleshs_GPU_NO_L2_Cache.txt", "a");
    

    //printf("Enter filter radius : ");
    //scanf("%d", &filter_radius);
    filter_radius = 16;
    //fprintf(fp,"for filter radius %d  ", filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;
    fprintf(fp, "%d x %d", imageW, imageW);

    if (imageW > 4){
      block_Dim = 4;
      grid_Dim = imageW / 4;
    }
    else{
      block_Dim = imageW;
      grid_Dim = 1;
    }

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_apotelesmata = (float *)malloc(imageW * imageH * sizeof(float)); //gia na epistrepsw to apotelesma apo thn GPU
    h_apotelesmata2= (float *)malloc(imageW * imageH * sizeof(float));  //gia na epistrepsw ta apotelemsata apo thn GPU gia to tiled convolution

    h_tiled     = (float *)malloc(imageW * imageH * sizeof(float)); 

    if ((h_Filter == 0 )||(h_Input == 0) || (h_Buffer == 0) || (h_OutputCPU == 0) || (h_apotelesmata2 == 0) || (h_apotelesmata == 0)){
        printf("Failure in memory allocation\n");
        exit (0);
    }
    //Device allocation
    int size = imageW * imageH * sizeof(float);
    hipMalloc((void**)&dev_Filter, FILTER_LENGTH * sizeof(float));
    hipMalloc((void**)&dev_Input, size);
    hipMalloc((void**)&dev_Buffer, size);
    hipMalloc((void**)&dev_new_Buffer, size);  //gia tis nees synarthseis pou tha dhmiourghsw
    hipMalloc((void**)&dev_OutputGPU, size);   
    hipMalloc((void**)&dev_new_OutputGPU, size); //gia tis nees synarthseis pou tha dhmiourghsw

    if ((dev_Filter == 0 )||(dev_Input == 0) || (dev_Buffer == 0) || (dev_OutputGPU == 0) || (dev_new_Buffer == 0) || (dev_new_OutputGPU == 0)){
        printf("Failure in Gpu memory allocation\n");
        exit (0);
    }

    //arxikopoihsh gia kathe endexomeno
    hipMemset(dev_Filter, 0 , FILTER_LENGTH * sizeof(float));
    hipMemset(dev_Input, 0, size);
    hipMemset(dev_Buffer, 0, size);
    hipMemset(dev_OutputGPU, 0, size);
    hipMemset(dev_new_OutputGPU, 0, size);
    hipMemset(dev_new_Buffer, 0, size);

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }
    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.

    //printf("CPU computation...\n");

    //start = clock();
    //convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    //convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
    //end = clock();
    //cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    //for (i = 0; i < imageW * imageH; i++) {    // stamataw tis ektypwseis gt dn pairnei apeiro xrono mexri na tis deiksei oles
     //  printf("%f\n",h_OutputCPU[i]);
    //}
    
    //memcpy afou gemisoun oi pinakes(eikones)
    

    
    hipMemcpy(dev_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_Input, h_Input, size, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(Filter_in_constant_mem),h_Filter,FILTER_LENGTH * sizeof(float));  //pernaw to Filtro sthn constant memory

    dim3 dimGrid(grid_Dim, grid_Dim);
    dim3 dimBlock(block_Dim, block_Dim);

    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas 

    //startGPU = clock();

    hipEventRecord(start, 0);
    printf("GPU computation...\n");
    convolutionRowGPU<<<dimGrid, dimBlock>>>(dev_Buffer, dev_Input, dev_Filter, imageW, imageH, filter_radius);

    // check for error dn vgainei gia na dw gt xtypaei to 2o error 
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
    }
    
    convolutionColumnGPU<<<dimGrid, dimBlock>>>(dev_OutputGPU, dev_Buffer, dev_Filter, imageW, imageH, filter_radius);

    // check for error
    hipError_t error1 = hipGetLastError();
    if(error1 != hipSuccess){
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error1));
    }

    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time_used_no_tile,start,stop);
    //endGPU = clock();
    hipMemcpy(h_apotelesmata, dev_OutputGPU , size, hipMemcpyDeviceToHost);


    //gpu_time_used = ((double) (endGPU - startGPU)) / CLOCKS_PER_SEC;
    
    //for (i = 0; i < imageW * imageH; i++) {
      //printf("%f\n",h_apotelesmata[i]);
    //}
    

    //float diafora = 0;
    //float Max_acc = -1;
  	//for (i = 0; i < imageW * imageH; i++) {
     // diafora = h_OutputCPU[i] - h_apotelesmata[i];
     // diafora = ABS(diafora);
     // if (diafora > Max_acc){
      //	Max_acc = diafora;
      //}
    //}

    dim3 new_block(block_Dim,block_Dim);
    dim3 new_grid(imageW/block_Dim, imageH/block_Dim);

    hipEventRecord(start1, 0);

    convolutionTileRowGPU<<<new_grid, new_block>>>(dev_new_Buffer, dev_Input, imageW, imageH, filter_radius);


    hipMemcpy(h_tiled, dev_new_Buffer , size, hipMemcpyDeviceToHost);



    convolutionTileColumnGPU<<<new_grid, new_block>>>(dev_new_OutputGPU, dev_new_Buffer, imageW, imageH, filter_radius);

    // check for error
    hipError_t error2 = hipGetLastError();
    if(error2 != hipSuccess){
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error2));
    }


    hipEventRecord(stop1, 0); 
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&gpu_time_used,start1,stop1);

    hipMemcpy(h_apotelesmata2, dev_new_OutputGPU , size, hipMemcpyDeviceToHost);



    for(i=0;i<imageW*imageH;i++) {
      printf("%f\n", h_tiled[i]);
    }

    int thesh = 0;
    float diafora = 0;
    float Max_acc = -1;
    for (i = 0; i < imageW * imageH; i++) {
      diafora = h_apotelesmata[i] - h_apotelesmata2[i];
      diafora = ABS(diafora);
      //printf("h_apotelesmata = %f kai h_apotelesmata2 = %f\n", h_apotelesmata[i], h_apotelesmata2[i]);
      if (diafora > Max_acc){
        Max_acc = diafora;
        thesh = i;
      }
    }

    printf ("H Megisth diafora metaksi tou tiledConvolution kai tou aplou einai %f kai einia sth thesh %d \n", Max_acc, thesh);
    printf("h_apotelesmata = %f kai h_apotelesmata2 = %f\n", h_apotelesmata[thesh], h_apotelesmata2[thesh]);

    printf ("to grid einia %d kai to block einia %d\n", grid_Dim, block_Dim);
    printf ("to neo grid einia %d kai to block einia %d\n", imageW/block_Dim, block_Dim);
    //printf ("The Max_accuracy is %f\n", Max_acc);
    //fprintf(fp,"the Max accuracy is    %32f\n",Max_acc);
    //printf("CPU time used in seconds is %f\n", cpu_time_used);
    printf("GPU time used in seconds is %f and GPU time for tiled version is %f\n", gpu_time_used_no_tile, gpu_time_used);
    fprintf(fp, "    %f   %f\n ", gpu_time_used_no_tile, gpu_time_used);
    
    //if(Max_acc > accuracy){
    	//printf("CPU and GPU results don't match and program is gonna terminate\n");
      //	exit(0);
    //}

    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    free(h_apotelesmata);
    hipFree(dev_Filter);
    hipFree(dev_Input);
    hipFree(dev_Buffer);
    hipFree(dev_OutputGPU);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}

