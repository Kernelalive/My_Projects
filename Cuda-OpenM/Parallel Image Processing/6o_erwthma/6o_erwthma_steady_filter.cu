/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005 

 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;		//edw einai to pou tha paei to filtro gi auto elegxei apo katw kai an to d einia ektos oriwn eikonas

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}

__global__ void convolutionRowGPU(double *h_Dst, double *h_Src, double *h_Filter, int imageW, int imageH, int filterR){

  int k;
  double sum = 0;
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  for (k = -filterR; k <= filterR; k++) {
    int d = ix + k;    //edw einai to pou tha paei to filtro gi auto elegxei apo katw kai an to d einia ektos oriwn eikonas
    if (d >= 0 && d < imageW) {
      sum += h_Src[iy * imageW + d] * h_Filter[filterR - k];
    }
    h_Dst[iy * imageW + ix] = sum;
  }
}


__global__ void convolutionColumnGPU(double *h_Dst, double *h_Src, double *h_Filter, int imageW, int imageH, int filterR){
	int k;
	double sum = 0;
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
  for (k = -filterR; k <= filterR; k++) {
    int d = iy + k;
    if (d >= 0 && d < imageH) {
      sum += h_Src[d * imageW + ix] * h_Filter[filterR - k];
    }
    h_Dst[iy * imageW + ix] = sum;
  }
}



////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    double
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *dev_Filter,
    *dev_Input,
    *dev_Buffer,
    *dev_OutputGPU,
    *h_apotelesmata;

    clock_t start, startGPU, endGPU, end;
    double cpu_time_used,gpu_time_used;


    int imageW;
    int imageH;
    int block_Dim, grid_Dim;
    unsigned int i;
    FILE * fp;
    FILE * fp1;

    fp = fopen("Xronoi_Ekteleshs_CPU_double_steady_filter.txt", "a");
    fp1 = fopen("Xronoi_Ekteleshs_GPU_double_steady_filter.txt", "a");
    

    //printf("Enter filter radius : ");
    //scanf("%d", &filter_radius);
    //fprintf(fp,"%d\n", filter_radius);
    //fprintf(fp1,"%d\n", filter_radius);
    filter_radius = 16;
    printf("filter radius is %d\n", filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    if (imageW > 32){
      block_Dim = 32;
      grid_Dim = imageW / 32;
    }
    else{
      block_Dim = imageW;
      grid_Dim = 1;
    }

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    fprintf(fp,"%d * %d\n", imageW, imageH);
    fprintf(fp1,"%d * %d\n", imageW, imageH);

    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (double *)malloc(FILTER_LENGTH * sizeof(double));
    h_Input     = (double *)malloc(imageW * imageH * sizeof(double));
    h_Buffer    = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputCPU = (double *)malloc(imageW * imageH * sizeof(double));
    h_apotelesmata = (double *)malloc(imageW * imageH * sizeof(double)); //gia na epistrepsw to apotelesma apo thn GPU

    if ((h_Filter == 0 )||(h_Input == 0) || (h_Buffer == 0) || (h_OutputCPU == 0)){
        printf("Failure in memory allocation\n");
        exit (0);
    }
    //Device allocation
    int size = imageW * imageH * sizeof(double);
    hipMalloc((void**)&dev_Filter, FILTER_LENGTH * sizeof(double));
    hipMalloc((void**)&dev_Input, size);
    hipMalloc((void**)&dev_Buffer, size);
    hipMalloc((void**)&dev_OutputGPU, size);

    if ((dev_Filter == 0 )||(dev_Input == 0) || (dev_Buffer == 0) || (dev_OutputGPU == 0)){
        printf("Failure in Gpu memory allocation\n");
        exit (0);
    }

    //arxikopoihsh gia kathe endexomeno
    hipMemset(dev_Filter, 0 , FILTER_LENGTH * sizeof(double));
    hipMemset(dev_Input, 0, size);
    hipMemset(dev_Buffer, 0, size);
    hipMemset(dev_OutputGPU, 0, size);

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (double)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
    }
    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.

    printf("CPU computation...\n");

    start = clock();
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    //for (i = 0; i < imageW * imageH; i++) {    // stamataw tis ektypwseis gt dn pairnei apeiro xrono mexri na tis deiksei oles
     //  printf("%f\n",h_OutputCPU[i]);
    //}
    
    //memcpy afou gemisoun oi pinakes(eikones)
    

    startGPU = clock();
    hipMemcpy(dev_Filter, h_Filter, FILTER_LENGTH * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_Input, h_Input, size, hipMemcpyHostToDevice);

    dim3 dimGrid(grid_Dim, grid_Dim);
    dim3 dimBlock(block_Dim, block_Dim);

    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas 

    printf("GPU computation...\n");
    convolutionRowGPU<<<dimGrid, dimBlock>>>(dev_Buffer, dev_Input, dev_Filter, imageW, imageH, filter_radius);

    // check for error dn vgainei gia na dw gt xtypaei to 2o error 
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
    }
    
    convolutionColumnGPU<<<dimGrid, dimBlock>>>(dev_OutputGPU, dev_Buffer, dev_Filter, imageW, imageH, filter_radius);

    // check for error
    hipError_t error1 = hipGetLastError();
    if(error1 != hipSuccess){
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error1));
    }
    
    hipMemcpy(h_apotelesmata, dev_OutputGPU , size, hipMemcpyDeviceToHost);

    endGPU = clock();

    gpu_time_used = ((double) (endGPU - startGPU)) / CLOCKS_PER_SEC;
    
    //for (i = 0; i < imageW * imageH; i++) {
    //  printf("%f\n",h_apotelesmata[i]);
    //}
    

    double diafora = 0;
    double Max_acc = -1;
  	for (i = 0; i < imageW * imageH; i++) {
      diafora = h_OutputCPU[i] - h_apotelesmata[i];
      diafora = ABS(diafora);
      if (diafora > Max_acc){
      	Max_acc = diafora;
      }
    }
    printf ("to grid einia %d kai to block einia %d\n", grid_Dim, block_Dim);
    printf ("The Max_accuracy is %f\n", Max_acc);
    //fprintf(fp,"the Max accuracy is    %32f\n",Max_acc);
    printf("CPU time used in seconds is %f\n", cpu_time_used);
    fprintf(fp,"%f\n", cpu_time_used);
    fprintf(fp1,"%f\n", gpu_time_used);
    printf("GPU time used in seconds is %f\n", gpu_time_used);
    
    if(Max_acc > accuracy){
    	printf("CPU and GPU results don't match and program is gonna terminate\n");
      	exit(0);
    }

    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    free(h_apotelesmata);
    hipFree(dev_Filter);
    hipFree(dev_Input);
    hipFree(dev_Buffer);
    hipFree(dev_OutputGPU);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}

