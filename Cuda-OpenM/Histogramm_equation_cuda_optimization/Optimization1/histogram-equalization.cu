#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"


__global__ void calc_histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin){
    
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    const int gridW = gridDim.x * blockDim.x;
    int img_position = iy * gridW + ix; //thesh mesa sthn eikona synarthsh tou gridW 

   // __shared__ hist_out[nbr_bin];

    if (img_position < nbr_bin) { //allagh tou img_position me threadIdx.x se auto to if anevazei poly to contrast
        hist_out[img_position] = 0;
    }

    __syncthreads();

    if(img_position < img_size){
        atomicAdd(&hist_out[img_in[img_position]],1);
    }
    __syncthreads();
}
 //int *lut = (int *)malloc(sizeof(int)*nbr_bin);
__global__ void calc_lut(int *lut, int * hist_in, int img_size, int nbr_bin){ 

    
    __shared__ int shared_hist[256];
    shared_hist[threadIdx.x] = hist_in[threadIdx.x];
    __syncthreads();
    __shared__  int cdf[256];
    __syncthreads();

    int i, min, d; 
    //int cdf = 0;
    min = 0;
    i = 0;

    while(min == 0){
        min = shared_hist[i++];
    }
    d = img_size - min;
    for (unsigned int stride = 1; stride <= threadIdx.x; stride *= 2) {
        __syncthreads();
        shared_hist[threadIdx.x] += shared_hist[threadIdx.x-stride];
    }
    cdf[threadIdx.x] = shared_hist[threadIdx.x];
    //printf("cdf = %d\n",cdf);
    __syncthreads();
    
	

    //for(i = 0; i <= threadIdx.x; i ++){	//tha mporouse na ginei me prefix sum san veltistoipohsh FIXME
      //  cdf += shared_hist[i];
      //  lut[i] = (cdf - min)*(nbr_bin - 1)/d;        
    //}
    //printf("cdf = %d\n",cdf);
    

    lut[threadIdx.x] = (int)(((float)cdf[threadIdx.x] - min)*255/d + 0.5);
    if(lut[threadIdx.x] < 0){
        lut[threadIdx.x] = 0;
    }
}
__global__ void calc_output(unsigned char * img_out, unsigned char * img_in, int * lut, int img_size){
    /* Get the result image */
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    const int gridW = gridDim.x * blockDim.x;
    int img_position1 = iy * gridW + ix; //thesh mesa sthn eikona synarthsh tou gridW 
    /* Construct the LUT by calculating the CDF */
    __syncthreads();
    if(img_position1 < img_size){
        if(lut[img_in[img_position1]] > 255){
            img_out[img_position1] = 255;
        }
        else{
            img_out[img_position1] = (unsigned char)lut[img_in[img_position1]];
        }
        
    }
}
