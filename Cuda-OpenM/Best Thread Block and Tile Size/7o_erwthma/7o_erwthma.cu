/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius ;

#define FILTER_LENGTH 	(2 * 16 + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005 
#define TILE_WIDTH  32

 __constant__ double Filter_in_constant_mem[FILTER_LENGTH];  //pernaw to filtro sthn constant memory gt auto mono tha diavazetai

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;		//edw einai to pou tha paei to filtro gi auto elegxei apo katw kai an to d einia ektos oriwn eikonas

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}

__global__ void convolutionRowGPU(double *h_Dst, double *h_Src, double *h_Filter, int imageW, int imageH, int filterR){

  int k;
  double sum = 0;
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  for (k = -filterR; k <= filterR; k++) {
    int d = ix + k;    //edw einai to pou tha paei to filtro gi auto elegxei apo katw kai an to d einia ektos oriwn eikonas
    if (d >= 0 && d < imageW) {
      sum += h_Src[iy * imageW + d] * h_Filter[filterR - k];
    }
    h_Dst[iy * imageW + ix] = sum;
  }
}


__global__ void convolutionColumnGPU(double *h_Dst, double *h_Src, double *h_Filter, int imageW, int imageH, int filterR){
	int k;
	double sum = 0;
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
  for (k = -filterR; k <= filterR; k++) {
    int d = iy + k;
    if (d >= 0 && d < imageH) {
      sum += h_Src[d * imageW + ix] * h_Filter[filterR - k];
    }
    h_Dst[iy * imageW + ix] = sum;
  }
}



__global__ void convolutionTileRowGPU(double *h_Dst, double *h_Src, int imageW, int imageH, int filterR){


  __shared__ double tiled_Src[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int k;

  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;

  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;

  tiled_Src[ty][tx] = h_Src[Row * imageW + (bx * TILE_WIDTH + tx)];
  __syncthreads(); //ola ta nhmata tou block exoun oloklhrwsei thn fortwsh ston tiled_Src

  double sum = 0;

  for (k = -filterR; k <= filterR; k++) {
    int d = tx + k;  

    if (d >= 0 && d < TILE_WIDTH ) {
      sum += tiled_Src[ty][d] * Filter_in_constant_mem[filterR - k];
    }
    else if((ix + k > 0) && (ix + k < imageW)){
      sum += h_Src[iy * imageW + (ix + k)] * Filter_in_constant_mem[filterR - k];
    }
    __syncthreads(); //gia na eksasfalisei oti ta nhmata tou block exoun teliwsei me ton tiled_Src

    h_Dst[Row * imageW + Col] = sum;
    __syncthreads();
  }
  __syncthreads();
}



__global__ void convolutionTileColumnGPU(double *h_Dst, double *h_Src, int imageW, int imageH, int filterR){


  __shared__ double tiled_Src[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int k;

  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;

  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;

  tiled_Src[ty][tx] = h_Src[(by * TILE_WIDTH + ty) * imageW + Col];
  __syncthreads(); //ola ta nhmata tou block exoun oloklhrwsei thn fortwsh ston tiled_Src

  double sum = 0;

  for (k = -filterR; k <= filterR; k++) {
    int d = ty + k;  

    if (d >= 0 && d < TILE_WIDTH ) {
      sum += tiled_Src[d][tx] * Filter_in_constant_mem[filterR - k];
    }
    else if((iy + k > 0) && (iy + k < imageW)){
      sum += h_Src[(iy + k) * imageW + ix] * Filter_in_constant_mem[filterR - k];
    }
    __syncthreads(); //gia na eksasfalisei oti ta nhmata tou block exoun teliwsei me ton tiled_Src
    h_Dst[Row * imageW + Col] = sum;
    __syncthreads();
  }

__syncthreads();
}



////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    double
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *dev_Filter,
    *dev_Input,
    *dev_Buffer,
    *dev_new_Buffer,
    *dev_OutputGPU,
    *dev_new_OutputGPU,
    *h_apotelesmata,
    *h_apotelesmata2;

    //clock_t startCPU, startGPU, endGPU, endCPU;
    double cpu_time_used;

    float gpu_time_used_no_tile, gpu_time_used;

    hipEvent_t start, stop, start1, stop1;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&start1);
    hipEventCreate(&stop1);


    int imageW;
    int imageH;
    int block_Dim, grid_Dim;
    unsigned int i;
    FILE * fp;
    fp = fopen("Xronoi_Ekteleshs_GPU_tiled_DOUBLE.txt", "a");
    

    //printf("Enter filter radius : ");
    //scanf("%d", &filter_radius);
    filter_radius = 16;
    //fprintf(fp,"for filter radius %d  ", filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;
    fprintf(fp, "%d x %d", imageW, imageW);

    if (imageW > 32){
      block_Dim = 32;
      grid_Dim = imageW / block_Dim;
    }
    else{
      block_Dim = imageW;
      grid_Dim = 1;
    }

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (double *)malloc(FILTER_LENGTH * sizeof(double));
    h_Input     = (double *)malloc(imageW * imageH * sizeof(double));
    h_Buffer    = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputCPU = (double *)malloc(imageW * imageH * sizeof(double));
    h_apotelesmata = (double *)malloc(imageW * imageH * sizeof(double)); //gia na epistrepsw to apotelesma apo thn GPU
    h_apotelesmata2= (double *)malloc(imageW * imageH * sizeof(double));  //gia na epistrepsw ta apotelemsata apo thn GPU gia to tiled convolution

    if ((h_Filter == 0 )||(h_Input == 0) || (h_Buffer == 0) || (h_OutputCPU == 0) || (h_apotelesmata2 == 0) || (h_apotelesmata == 0)){
        printf("Failure in memory allocation\n");
        exit (0);
    }
    //Device allocation
    int size = imageW * imageH * sizeof(double);
    hipMalloc((void**)&dev_Filter, FILTER_LENGTH * sizeof(double));
    hipMalloc((void**)&dev_Input, size);
    hipMalloc((void**)&dev_Buffer, size);
    hipMalloc((void**)&dev_new_Buffer, size);  //gia tis nees synarthseis pou tha dhmiourghsw
    hipMalloc((void**)&dev_OutputGPU, size);   
    hipMalloc((void**)&dev_new_OutputGPU, size); //gia tis nees synarthseis pou tha dhmiourghsw

    if ((dev_Filter == 0 )||(dev_Input == 0) || (dev_Buffer == 0) || (dev_OutputGPU == 0) || (dev_new_Buffer == 0) || (dev_new_OutputGPU == 0)){
        printf("Failure in Gpu memory allocation\n");
        exit (0);
    }

    //arxikopoihsh gia kathe endexomeno
    hipMemset(dev_Filter, 0 , FILTER_LENGTH * sizeof(double));
    hipMemset(dev_Input, 0, size);
    hipMemset(dev_Buffer, 0, size);
    hipMemset(dev_OutputGPU, 0, size);
    hipMemset(dev_new_OutputGPU, 0, size);
    hipMemset(dev_new_Buffer, 0, size);

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (double)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
    }
    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.

    //printf("CPU computation...\n");

    //start = clock();
    //convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    //convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
    //end = clock();
    //cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    //for (i = 0; i < imageW * imageH; i++) {    // stamataw tis ektypwseis gt dn pairnei apeiro xrono mexri na tis deiksei oles
     //  printf("%f\n",h_OutputCPU[i]);
    //}
    
    //memcpy afou gemisoun oi pinakes(eikones)
    

    
    hipMemcpy(dev_Filter, h_Filter, FILTER_LENGTH * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_Input, h_Input, size, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(Filter_in_constant_mem),h_Filter,FILTER_LENGTH * sizeof(double));  //pernaw to Filtro sthn constant memory

    dim3 dimGrid(grid_Dim, grid_Dim);
    dim3 dimBlock(block_Dim, block_Dim);

    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas 

    //startGPU = clock();
    hipEventRecord(start, 0);
    printf("GPU computation...\n");
    convolutionRowGPU<<<dimGrid, dimBlock>>>(dev_Buffer, dev_Input, dev_Filter, imageW, imageH, filter_radius);

    // check for error dn vgainei gia na dw gt xtypaei to 2o error 
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
    }
    
    convolutionColumnGPU<<<dimGrid, dimBlock>>>(dev_OutputGPU, dev_Buffer, dev_Filter, imageW, imageH, filter_radius);

    // check for error
    hipError_t error1 = hipGetLastError();
    if(error1 != hipSuccess){
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error1));
    }
    
    //endGPU = clock();
    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time_used_no_tile,start,stop);

    hipMemcpy(h_apotelesmata, dev_OutputGPU , size, hipMemcpyDeviceToHost);


    //gpu_time_used = ((double) (endGPU - startGPU)) / CLOCKS_PER_SEC;
    
    //for (i = 0; i < imageW * imageH; i++) {
      //printf("%f\n",h_apotelesmata[i]);
    //}
    

    //double diafora = 0;
    //double Max_acc = -1;
  	//for (i = 0; i < imageW * imageH; i++) {
     // diafora = h_OutputCPU[i] - h_apotelesmata[i];
     // diafora = ABS(diafora);
     // if (diafora > Max_acc){
      //	Max_acc = diafora;
      //}
    //}

    dim3 new_block(block_Dim,block_Dim);
    dim3 new_grid(imageW/block_Dim,imageH/block_Dim);

    hipEventRecord(start1, 0);

    convolutionTileRowGPU<<<new_grid, new_block>>>(dev_new_Buffer, dev_Input, imageW, imageH, filter_radius);
    convolutionTileColumnGPU<<<new_grid, new_block>>>(dev_new_OutputGPU, dev_new_Buffer, imageW, imageH, filter_radius);

    hipEventRecord(stop1, 0); 
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&gpu_time_used,start1,stop1);

    hipMemcpy(h_apotelesmata2, dev_new_OutputGPU , size, hipMemcpyDeviceToHost);

    int thesh = 0;
    double diafora = 0;
    double Max_acc = -1;
    for (i = 0; i < imageW * imageH; i++) {
      diafora = h_apotelesmata[i] - h_apotelesmata2[i];
      diafora = ABS(diafora);
      //printf("h_apotelesmata = %f kai h_apotelesmata2 = %f\n", h_apotelesmata[i], h_apotelesmata2[i]);
      if (diafora > Max_acc){
        Max_acc = diafora;
        thesh = i;
      }
    }

    printf ("H Megisth diafora metaksi tou tiledConvolution kai tou aplou einai %f kai einia sth thesh %d \n", Max_acc, thesh);


    printf ("to grid einia %d kai to block einia %d\n", grid_Dim, block_Dim);
    //printf ("The Max_accuracy is %f\n", Max_acc);
    //fprintf(fp,"the Max accuracy is    %32f\n",Max_acc);
    //printf("CPU time used in seconds is %f\n", cpu_time_used);
    printf("GPU time used in seconds is %f\n", gpu_time_used);
    fprintf(fp, "    %f\n", gpu_time_used);
    
    //if(Max_acc > accuracy){
    	//printf("CPU and GPU results don't match and program is gonna terminate\n");
      //	exit(0);
    //}

    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);
    free(h_apotelesmata);
    hipFree(dev_Filter);
    hipFree(dev_Input);
    hipFree(dev_Buffer);
    hipFree(dev_OutputGPU);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}

